#include <hip/hip_runtime.h>

template<class Func> __global__ void invoke(Func func) { func(); }

int main() {
    int* a = nullptr;
    hipMalloc(&a, sizeof(int));
    invoke<<<1, 1>>>(
        [=] __device__ () {
            a[threadIdx.x] = 1;
        }
    );
}
